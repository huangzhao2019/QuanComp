#include <hip/hip_runtime.h>
#include <cassert>

#if defined(FEATURE_BLS12_381)
# include <ff/bls12-381.hpp>
#else
# error "no FEATURE"
#endif

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_inf_t affine_t;
typedef fr_t scalar_t;

#include <iostream>
#include <fstream>
#include <tuple>
using namespace std;

tuple<scalar_t*, affine_t*> readData(size_t npoints) {
    ifstream file_obj;
    scalar_t* scalars = new scalar_t[npoints];
    affine_t* points = new affine_t[npoints];
    string fname = "./data/affine_t_" + to_string(npoints) + ".dat";
    file_obj.open(fname, ios::out | ios::binary);
    for (unsigned i = 0; i < npoints; i++) {
        // file_obj.read((char*)&scalars[i], sizeof(scalars[i]));
        file_obj.read((char*)&points[i], sizeof(points[i]));
    }
    file_obj.close();
    return make_tuple(scalars, points);
}

// int main() {
//     size_t npoints = 1000;
//     scalar_t* scalars = new scalar_t[npoints];
//     affine_t* points = new affine_t[npoints];
//     tie(scalars, points) = readData(npoints);
//     printf("read data done\n");
// }


int main() {
    size_t npoints = 1000;
    scalar_t* scalars = new scalar_t[npoints];
    affine_t* points = new affine_t[npoints];
    tie(scalars, points) = readData(npoints);
    printf("Testing curve on host...");
    for (size_t i = 0; i<npoints; i++) {
        affine_t p = points[i];
        // enable accessors in the source code to make the below lines work
        
        fp_t x = p.return_x();
        fp_t y = p.return_y();


        fp_t const1 = fp_t::one();
        fp_t r = (y*y)-(x*x*x)-(const1+const1+const1+const1);

        assert( r.is_zero() );
    }
}
